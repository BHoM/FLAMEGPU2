#include "hip/hip_runtime.h"
#include "flamegpu/runtime/messaging/MessageBruteForceSorted/MessageBruteForceSortedHost.h"
#include "flamegpu/runtime/messaging/MessageBruteForceSorted/MessageBruteForceSortedDevice.cuh"

#include "flamegpu/gpu/CUDAScatter.cuh"
#ifdef _MSC_VER
#pragma warning(push, 1)
#pragma warning(disable : 4706 4834)
#include <hipcub/hipcub.hpp>
#pragma warning(pop)
#else
#include <hipcub/hipcub.hpp>
#endif


namespace flamegpu {


MessageBruteForceSorted::CUDAModelHandler::CUDAModelHandler(CUDAMessage &a)
  : MessageSpecialisationHandler()
  , sim_message(a) {
    NVTX_RANGE("Spatial3D::CUDAModelHandler");
    const Data &d = (const Data &)a.getMessageDescription();
    hd_data.radius = d.radius;
    hd_data.min[0] = d.minX;
    hd_data.min[1] = d.minY;
    hd_data.min[2] = d.minZ;
    hd_data.max[0] = d.maxX;
    hd_data.max[1] = d.maxY;
    hd_data.max[2] = d.maxZ;
    binCount = 1;
    for (unsigned int axis = 0; axis < 3; ++axis) {
        hd_data.environmentWidth[axis] = hd_data.max[axis] - hd_data.min[axis];
        hd_data.gridDim[axis] = static_cast<unsigned int>(ceil(hd_data.environmentWidth[axis] / hd_data.radius));
        binCount *= hd_data.gridDim[axis];
    }
    // Device allocation occurs in allocateMetaDataDevicePtr rather than the constructor.
}

__global__ void computeSpatialHash(
    const MessageBruteForceSorted::MetaData *md,
    unsigned int* bin_index,
    unsigned int* tid,
    unsigned int message_count,
    const float * __restrict__ x,
    const float * __restrict__ y,
    const float * __restrict__ z) {
    unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    // Kill excess threads
    if (index >= message_count) return;

    MessageBruteForceSorted::GridPos3D gridPos = getGridPosition3D(md, x[index], y[index], z[index]);
    unsigned int hash = getHash3D(md, gridPos);
    bin_index[index] = hash;
    tid[index] = index;
}

void MessageBruteForceSorted::CUDAModelHandler::init(CUDAScatter &, const unsigned int &) {
    allocateMetaDataDevicePtr();
    // Allocate messages
    hd_data.length = 0;  // This value should already be 0
    gpuErrchk(hipMemcpy(d_data, &hd_data, sizeof(MetaData), hipMemcpyHostToDevice));
}

void MessageBruteForceSorted::CUDAModelHandler::allocateMetaDataDevicePtr() {
    if (d_data == nullptr) {
        gpuErrchk(hipMalloc(&d_data, sizeof(MetaData)));
        gpuErrchk(hipMemcpy(d_data, &hd_data, sizeof(MetaData), hipMemcpyHostToDevice));
        resizeCubTemp();
    }
}

void MessageBruteForceSorted::CUDAModelHandler::freeMetaDataDevicePtr() {
    if (d_data != nullptr) {
        d_CUB_temp_storage_bytes = 0;
        gpuErrchk(hipFree(d_CUB_temp_storage));
        gpuErrchk(hipFree(d_data));
        d_CUB_temp_storage = nullptr;
        d_data = nullptr;
        if (d_keys) {
            d_keys_vals_storage_bytes = 0;
            gpuErrchk(hipFree(d_keys));
            gpuErrchk(hipFree(d_vals));
            gpuErrchk(hipFree(d_keys_out));
            gpuErrchk(hipFree(d_vals_out));
            d_keys = nullptr;
            d_vals = nullptr;
            d_keys_out = nullptr;
            d_vals_out = nullptr;
        }
    }
}

void MessageBruteForceSorted::CUDAModelHandler::buildIndex(CUDAScatter &scatter, const unsigned int &streamId, const hipStream_t &stream) {
    NVTX_RANGE("MessageBruteForceSorted::CUDAModelHandler::buildIndex");
    
    unsigned int MESSAGE_COUNT = this->sim_message.getMessageCount();
    if (MESSAGE_COUNT != hd_data.length) {
        hd_data.length = MESSAGE_COUNT;
        gpuErrchk(hipMemcpy(d_data, &hd_data, sizeof(MetaData), hipMemcpyHostToDevice));
        
        // Resize temp storage
    }
    resizeKeysVals(this->sim_message.getMaximumListSize());  // Resize based on allocated amount rather than message count
    
    { 
        int blockSize;  // The launch configurator returned block size
        gpuErrchk(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blockSize, computeSpatialHash, 32, 0));  // Randomly 32
                                                                                                         // Round up according to array size
        int gridSize = (MESSAGE_COUNT + blockSize - 1) / blockSize;
        computeSpatialHash <<<gridSize, blockSize, 0, stream >>>(d_data, d_keys, d_vals, MESSAGE_COUNT,
            reinterpret_cast<float*>(this->sim_message.getReadPtr("x")),
            reinterpret_cast<float*>(this->sim_message.getReadPtr("y")),
            reinterpret_cast<float*>(this->sim_message.getReadPtr("z")));
    }
    {  // Reorder messages

        // d_keys currently contains hashes, d_vals contains tid 1..message_count
        // Perform pair sort on hashes to get sorted message order
        gpuErrchk(hipcub::DeviceRadixSort::SortPairs(d_CUB_temp_storage, d_CUB_temp_storage_bytes, d_keys, d_keys_out, d_vals, d_vals_out, MESSAGE_COUNT));
       
        // Reorder actual messages
        // Copy messages from d_messages to d_messages_swap, in hash order
        //scatter.pbm_reorder(streamId, stream, this->sim_message.getMessageDescription().variables, this->sim_message.getReadList(), this->sim_message.getWriteList(), MESSAGE_COUNT, d_keys, d_vals, hd_data.PBM);
        //this->sim_message.swap();  // Stream id is unused here
        //gpuErrchk(hipStreamSynchronize(stream));  // Not striclty neceesary while pbm_reorder is synchronous.
    }
}

void MessageBruteForceSorted::CUDAModelHandler::resizeCubTemp() {
}

void MessageBruteForceSorted::CUDAModelHandler::resizeKeysVals(const unsigned int &newSize) {
    size_t bytesCheck = newSize * sizeof(unsigned int);
    if (bytesCheck > d_keys_vals_storage_bytes) {
        if (d_keys) {
            gpuErrchk(hipFree(d_keys));
            gpuErrchk(hipFree(d_vals));
            gpuErrchk(hipFree(d_keys_out));
            gpuErrchk(hipFree(d_vals_out));
        }
        d_keys_vals_storage_bytes = bytesCheck;
        gpuErrchk(hipMalloc(&d_keys, d_keys_vals_storage_bytes));
        gpuErrchk(hipMalloc(&d_vals, d_keys_vals_storage_bytes));
        gpuErrchk(hipMalloc(&d_keys_out, d_keys_vals_storage_bytes));
        gpuErrchk(hipMalloc(&d_vals_out, d_keys_vals_storage_bytes));
        gpuErrchk(hipcub::DeviceRadixSort::SortPairs(nullptr, d_CUB_temp_storage_bytes, d_keys_in, d_keys_out, d_vals_in, d_vals_out, newSize));
    }
}

MessageBruteForceSorted::Data::Data(const std::shared_ptr<const ModelData> &model, const std::string &message_name)
    : MessageSpatial2D::Data(model, message_name)
    , minZ(NAN)
    , maxZ(NAN) {
    description = std::unique_ptr<Description>(new Description(model, this));
    description->newVariable<float>("z");
}
MessageBruteForceSorted::Data::Data(const std::shared_ptr<const ModelData> &model, const Data &other)
    : MessageSpatial2D::Data(model, other)
    , minZ(other.minZ)
    , maxZ(other.maxZ) {
    description = std::unique_ptr<Description>(model ? new Description(model, this) : nullptr);
    if (isnan(minZ)) {
        THROW exception::InvalidMessage("Environment minimum z bound has not been set in spatial message '%s'\n", other.name.c_str());
    }
    if (isnan(maxZ)) {
        THROW exception::InvalidMessage("Environment maximum z bound has not been set in spatial message '%s'\n", other.name.c_str());
    }
}
MessageBruteForceSorted::Data *MessageBruteForceSorted::Data::clone(const std::shared_ptr<const ModelData> &newParent) {
    return new Data(newParent, *this);
}
std::unique_ptr<MessageSpecialisationHandler> MessageBruteForceSorted::Data::getSpecialisationHander(CUDAMessage &owner) const {
    return std::unique_ptr<MessageSpecialisationHandler>(new CUDAModelHandler(owner));
}
std::type_index MessageBruteForceSorted::Data::getType() const { return std::type_index(typeid(MessageBruteForceSorted)); }

MessageBruteForceSorted::Description::Description(const std::shared_ptr<const ModelData> &_model, Data *const data)
    : MessageBruteForce::Description(_model, data) { }

void MessageBruteForceSorted::Description::setRadius(const float &r) {
    if (r <= 0) {
        THROW exception::InvalidArgument("Spatial messaging radius must be a positive value, %f is not valid.", r);
    }
    reinterpret_cast<Data *>(message)->radius = r;
}
void MessageBruteForceSorted::Description::setMinX(const float &x) {
    if (!isnan(reinterpret_cast<Data *>(message)->maxX) &&
        x >= reinterpret_cast<Data *>(message)->maxX) {
        THROW exception::InvalidArgument("Spatial messaging min x bound must be lower than max bound, %f !< %f", x, reinterpret_cast<Data *>(message)->maxX);
    }
    reinterpret_cast<Data *>(message)->minX = x;
}
void MessageBruteForceSorted::Description::setMinY(const float &y) {
    if (!isnan(reinterpret_cast<Data *>(message)->maxY) &&
        y >= reinterpret_cast<Data *>(message)->maxY) {
        THROW exception::InvalidArgument("Spatial messaging min bound must be lower than max bound, %f !< %f", y, reinterpret_cast<Data *>(message)->maxY);
    }
    reinterpret_cast<Data *>(message)->minY = y;
}
void MessageBruteForceSorted::Description::setMinZ(const float &z) {
    if (!isnan(reinterpret_cast<Data *>(message)->maxZ) &&
        z >= reinterpret_cast<Data *>(message)->maxZ) {
        THROW exception::InvalidArgument("Spatial messaging min z bound must be lower than max bound, %f !< %f", z, reinterpret_cast<Data *>(message)->maxZ);
    }
    reinterpret_cast<Data *>(message)->minZ = z;
}
void MessageBruteForceSorted::Description::setMin(const float &x, const float &y, const float &z) {
    if (!isnan(reinterpret_cast<Data *>(message)->maxX) &&
        x >= reinterpret_cast<Data *>(message)->maxX) {
        THROW exception::InvalidArgument("Spatial messaging min x bound must be lower than max bound, %f !< %f", x, reinterpret_cast<Data *>(message)->maxX);
    }
    if (!isnan(reinterpret_cast<Data *>(message)->maxY) &&
        y >= reinterpret_cast<Data *>(message)->maxY) {
        THROW exception::InvalidArgument("Spatial messaging min y bound must be lower than max bound, %f !< %f", y, reinterpret_cast<Data *>(message)->maxY);
    }
    if (!isnan(reinterpret_cast<Data *>(message)->maxZ) &&
        z >= reinterpret_cast<Data *>(message)->maxZ) {
        THROW exception::InvalidArgument("Spatial messaging min z bound must be lower than max bound, %f !< %f", z, reinterpret_cast<Data *>(message)->maxZ);
    }
    reinterpret_cast<Data *>(message)->minX = x;
    reinterpret_cast<Data *>(message)->minY = y;
    reinterpret_cast<Data *>(message)->minZ = z;
}
void MessageBruteForceSorted::Description::setMaxX(const float &x) {
    if (!isnan(reinterpret_cast<Data *>(message)->minX) &&
        x <= reinterpret_cast<Data *>(message)->minX) {
        THROW exception::InvalidArgument("Spatial messaging max x bound must be greater than min bound, %f !> %f", x, reinterpret_cast<Data *>(message)->minX);
    }
    reinterpret_cast<Data *>(message)->maxX = x;
}
void MessageBruteForceSorted::Description::setMaxY(const float &y) {
    if (!isnan(reinterpret_cast<Data *>(message)->minY) &&
        y <= reinterpret_cast<Data *>(message)->minY) {
        THROW exception::InvalidArgument("Spatial messaging max y bound must be greater than min bound, %f !> %f", y, reinterpret_cast<Data *>(message)->minY);
    }
    reinterpret_cast<Data *>(message)->maxY = y;
}
void MessageBruteForceSorted::Description::setMaxZ(const float &z) {
    if (!isnan(reinterpret_cast<Data *>(message)->minZ) &&
        z <= reinterpret_cast<Data *>(message)->minZ) {
        THROW exception::InvalidArgument("Spatial messaging max z bound must be greater than min bound, %f !> %f", z, reinterpret_cast<Data *>(message)->minZ);
    }
    reinterpret_cast<Data *>(message)->maxZ = z;
}
void MessageBruteForceSorted::Description::setMax(const float &x, const float &y, const float &z) {
    if (!isnan(reinterpret_cast<Data *>(message)->minX) &&
        x <= reinterpret_cast<Data *>(message)->minX) {
        THROW exception::InvalidArgument("Spatial messaging max x bound must be greater than min bound, %f !> %f", x, reinterpret_cast<Data *>(message)->minX);
    }
    if (!isnan(reinterpret_cast<Data *>(message)->minY) &&
        y <= reinterpret_cast<Data *>(message)->minY) {
        THROW exception::InvalidArgument("Spatial messaging max y bound must be greater than min bound, %f !> %f", y, reinterpret_cast<Data *>(message)->minY);
    }
    if (!isnan(reinterpret_cast<Data *>(message)->minZ) &&
        z <= reinterpret_cast<Data *>(message)->minZ) {
        THROW exception::InvalidArgument("Spatial messaging max z bound must be greater than min bound, %f !> %f", z, reinterpret_cast<Data *>(message)->minZ);
    }
    reinterpret_cast<Data *>(message)->maxX = x;
    reinterpret_cast<Data *>(message)->maxY = y;
    reinterpret_cast<Data *>(message)->maxZ = z;
}

float MessageBruteForceSorted::Description::getRadius() const {
    return reinterpret_cast<Data *>(message)->radius;
}
float MessageBruteForceSorted::Description::getMinX() const {
    return reinterpret_cast<Data *>(message)->minX;
}
float MessageBruteForceSorted::Description::getMinY() const {
    return reinterpret_cast<Data *>(message)->minY;
}
float MessageBruteForceSorted::Description::getMinZ() const {
    return reinterpret_cast<Data *>(message)->minZ;
}
float MessageBruteForceSorted::Description::getMaxX() const {
    return reinterpret_cast<Data *>(message)->maxX;
}
float MessageBruteForceSorted::Description::getMaxY() const {
    return reinterpret_cast<Data *>(message)->maxY;
}
float MessageBruteForceSorted::Description::getMaxZ() const {
    return reinterpret_cast<Data *>(message)->maxZ;
}

}  // namespace flamegpu

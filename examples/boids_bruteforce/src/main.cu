#include "hip/hip_runtime.h"
// Modified temporary example to demonstrate proof of concept of RTC cuJit patching

#include <hip/hip_runtime.h>
#include <random>
#include <memory>

#include "flamegpu/util/detail/compute_capability.cuh"

#ifdef _MSC_VER
#pragma warning(push, 2)
#include "jitify/jitify.hpp"
#pragma warning(pop)
#else
#include "jitify/jitify.hpp"
#endif

#if defined(_DEBUG) || defined(D_DEBUG)
#define CUDA_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define CUDA_CHECK(location) { gpuAssert(hipDeviceSynchronize(), __FILE__, __LINE__); }
#else
#define CUDA_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define CUDA_CHECK(location) { gpuAssert(hipPeekAtLastError(), __FILE__, __LINE__); }
#endif
inline void gpuAssert(hipError_t code, const char* file, int line) {
    if (code != hipSuccess) {
        if (line >= 0) {
            fprintf(stderr, "CUDA Error: %s(%d): %s", file, line, hipGetErrorString(code));
        }
        else {
            fprintf(stderr, "CUDA Error: %s(%d): %s", file, line, hipGetErrorString(code));
        }
        exit(EXIT_FAILURE);
    }
}

const char* test_kernel_src = R"###(
__device__ float input[1];
__global__ void test_patching(float *output, const size_t len) {
    for(int i = 0; i < len; ++i)
        output[i] = input[i];
}
)###";

int main(int argc, const char ** argv) {

    CUDA_CALL(hipFree(nullptr));

    // Allocate buffers
    const size_t INPUT_LEN = 100;
    float* d_input = nullptr, *d_output = nullptr;
    float *h_input = nullptr, *h_output = nullptr;
    CUDA_CALL(hipMalloc(&d_input, INPUT_LEN * sizeof(float)));
    CUDA_CALL(hipMalloc(&d_output, INPUT_LEN * sizeof(float)));
    h_input = (float*)malloc(INPUT_LEN * sizeof(float));
    h_output = (float*)malloc(INPUT_LEN * sizeof(float));

    // Fill buffer with random data
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(-1.0, 1.0);
    for (int i = 0; i < INPUT_LEN; ++i)
        h_input[i] = dis(gen);
    CUDA_CALL(hipMemcpy(d_input, h_input, INPUT_LEN * sizeof(float), hipMemcpyHostToDevice));

    // Compile the kernel
   
    // vector of compiler options for jitify
    std::vector<std::string> options;
    std::vector<std::string> headers;

    // Set the compilation architecture target if it was successfully detected.
    int currentDeviceIdx = 0;
    hipError_t status = hipGetDevice(&currentDeviceIdx);
    if (status == hipSuccess) {
        int arch = flamegpu::util::detail::compute_capability::getComputeCapability(currentDeviceIdx);
        options.push_back(std::string("--gpu-architecture=compute_" + std::to_string(arch)));
    }

    // jitify to create program (with compilation settings)
    std::unique_ptr<jitify::experimental::KernelInstantiation> kernel_instance;
    try {
        auto program = jitify::experimental::Program(test_kernel_src, headers, options);
        auto kernel = program.kernel("test_patching");
        kernel_instance = std::make_unique<jitify::experimental::KernelInstantiation>(kernel, std::vector<std::string>{});
    } catch (std::runtime_error const&) {
        fprintf(stderr, "Compilation failed, see stdout.\n");
        return EXIT_FAILURE;
    }

    // Serialise
    std::string serialized_kernel = kernel_instance->serialize();

    {
        // Deserialise with cujit options
        const unsigned int nopts = 3;
        hipJitOption opts[3] = { hipJitOptionGlobalSymbolCount, hipJitOptionGlobalSymbolNames, hipJitOptionGlobalSymbolAddresses };
        unsigned int SYMBOL_COUNT = 1;
        const char *SYMBOL_NAMES[1] = { "input" };
        void *SYMBOL_ADDRESSES[1] = { d_input };
        void *optvals[3] = { &SYMBOL_COUNT, SYMBOL_NAMES, SYMBOL_ADDRESSES };
        jitify::experimental::KernelInstantiation patched_kernel_instance =
        //jitify::experimental::KernelInstantiation::deserialize(serialized_kernel);
        jitify::experimental::KernelInstantiation::deserialize(serialized_kernel, nopts, opts, optvals);

        // Execute kernel
        hipError_t a = patched_kernel_instance.configure(1, 1).launch({
            reinterpret_cast<void*>(&h_output),
            const_cast<void*>(reinterpret_cast<const void*>(&INPUT_LEN))
        });
        if (a != hipError_t::hipSuccess) {
            const char* err_str = nullptr;
            hipDrvGetErrorString(a, &err_str);
            fprintf(stderr, "Executing instance 1 failed: %s\n", err_str);
            return EXIT_FAILURE;
        }
        CUDA_CHECK("Launch 1");
    }

    // Validate result
    CUDA_CALL(hipMemcpy(h_output, d_output, INPUT_LEN * sizeof(float), hipMemcpyDeviceToHost));
    unsigned int error_count = 0;
    for (int i = 0; i < INPUT_LEN; ++i)
        error_count += h_output[i] == h_input[i] ? 0 : 1;

    printf("Test 1 had %u errors!\n", error_count);

    // Deserialise with different cujit options

    // Execute kernel

    // Validate result

    return 0;
}


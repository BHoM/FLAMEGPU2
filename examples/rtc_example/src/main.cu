#include "hip/hip_runtime.h"
/******************************************************************************
 * main.cu is a host function that prepares data array and passes it to the CUDA kernel.
 * This main.cu would either be specified by a user or automatically generated from the model.xml.
 * Each of the API functions will have a 121 mapping with XML elements
 * The API is very similar to FLAME 2. The directory structure and general project is set out very similarly.

 * Single Agent model example

 ******************************************************************************
 * Author  Paul Richmond, Mozhgan Kabiri Chimeh
 * Date    Feb 2017
 *****************************************************************************/

#include "flamegpu/flame_api.h"


/* must be compiled separately using FLAME GPU builder
 * This will generate object files for different architecture targets as well as ptx info for each agent function (registers, memory use etc.)
 * http://stackoverflow.com/questions/12388207/interpreting-output-of-ptxas-options-v
 */

//#define AGENT_COUNT 32
#define EXPECT_EQ(x, y) if (x != y) printf("%d not equal to %d", static_cast<int>(x), static_cast<int>(y))


const char* out_optional2D = R"###(
FLAMEGPU_AGENT_FUNCTION(out_optional2D, MsgNone, MsgSpatial2D) {
    if (FLAMEGPU->getVariable<int>("do_output")) {
        FLAMEGPU->message_out.setVariable<int>("id", FLAMEGPU->getVariable<int>("id"));
        FLAMEGPU->message_out.setLocation(
            FLAMEGPU->getVariable<float>("x"),
            FLAMEGPU->getVariable<float>("y"));
    }
    return ALIVE;
}
)###";

const char* in2D = R"###(
FLAMEGPU_AGENT_FUNCTION(in2D, MsgSpatial2D, MsgNone) {
    const float x1 = FLAMEGPU->getVariable<float>("x");
    const float y1 = FLAMEGPU->getVariable<float>("y");
    unsigned int count = 0;
    unsigned int badCount = 0;
     unsigned int myBin[2] = {
         static_cast<unsigned int>(x1),
         static_cast<unsigned int>(y1)
     };
    // Count how many messages we received (including our own)
    // This is all those which fall within the 3x3x3 Moore neighbourhood
    // Not our search radius
    for (const auto &message : FLAMEGPU->message_in(x1, y1)) {
         unsigned int msgBin[2] = {
             static_cast<unsigned int>(message.getVariable<float>("x")),
             static_cast<unsigned int>(message.getVariable<float>("y"))
         };
         bool isBad = false;
         for (unsigned int i = 0; i < 2; ++i) {  // Iterate axis
             int binDiff = myBin[i] - msgBin[i];
             if (binDiff > 1 || binDiff < -1) {
                 isBad = true;
             }
         }
        count++;
        badCount = isBad ? badCount + 1 : badCount;
    }
    FLAMEGPU->setVariable<unsigned int>("count", count);
    FLAMEGPU->setVariable<unsigned int>("badCount", badCount);
    return ALIVE;
}
)###";


const unsigned int AGENT_COUNT = 128;





/**
 * Test an RTC function to an agent function condition (where the condition is not compiled using RTC)
 */
int main() {
    std::unordered_map<int, unsigned int> bin_counts;
    std::unordered_map<int, unsigned int> bin_counts_optional;
    // Construct model
    ModelDescription model("Spatial2DMsgTestModel");
    {   // Location message
        MsgSpatial2D::Description &message = model.newMessage<MsgSpatial2D>("location");
        message.setMin(0, 0);
        message.setMax(11, 11);
        message.setRadius(1);
        // 11x11 bins, total 121
        message.newVariable<int>("id");  // unused by current test
    }
       // Circle agent
        AgentDescription &agent = model.newAgent("agent");
        agent.newVariable<int>("id");
        agent.newVariable<float>("x");
        agent.newVariable<float>("y");
        agent.newVariable<int>("do_output");  // NEW!
        agent.newVariable<unsigned int>("myBin");  // This will be presumed bin index of the agent, might not use this
        agent.newVariable<unsigned int>("count");  // Store the distance moved here, for validation
        agent.newVariable<unsigned int>("badCount");  // Store how many messages are out of range
        auto &af = agent.newRTCFunction("out", out_optional2D);  // NEW!
        af.setMessageOutput("location");
        af.setMessageOutputOptional(true);  // NEW!
        auto& inf = agent.newRTCFunction("in", in2D);
        inf.setMessageInput("location");
    
       // Layer #1
        LayerDescription &l1 = model.newLayer();
        l1.addAgentFunction(af);  // NEW!
    
       // Layer #2
        LayerDescription &l2 = model.newLayer();
        l2.addAgentFunction(inf);
   
    CUDAAgentModel cuda_model(model);

    const int AGENT_COUNT = 2049;
    AgentPopulation population(model.Agent("agent"), AGENT_COUNT);
    // Initialise agents (TODO)
    {
        // Currently population has not been init, so generate an agent population on the fly
        std::default_random_engine rng;
        std::uniform_real_distribution<float> dist(0.0f, 11.0f);
        std::uniform_real_distribution<float> dist5(0.0f, 5.0f);
        for (unsigned int i = 0; i < AGENT_COUNT; i++) {
            AgentInstance instance = population.getNextInstance();
            instance.setVariable<int>("id", i);
            float pos[3] = { dist(rng), dist(rng), dist(rng) };
            int do_output = dist5(rng) < 4 ? 1 : 0;  // 80% chance of output  // NEW!
            instance.setVariable<float>("x", pos[0]);
            instance.setVariable<float>("y", pos[1]);
            instance.setVariable<int>("do_output", do_output);  // NEW!
            // Solve the bin index
            const unsigned int bin_pos[2] = {
                (unsigned int)(pos[0] / 1),
                (unsigned int)(pos[1] / 1)
            };
            const unsigned int bin_index =
                bin_pos[1] * 11 +
                bin_pos[0];
            instance.setVariable<unsigned int>("myBin", bin_index);
            // Create it if it doesn't already exist
            bin_counts[bin_index] += 1;
            if (do_output) {  // NEW!
                bin_counts_optional[bin_index] += 1;  // NEW!
            }
        }
        cuda_model.setPopulationData(population);
    }

    // Generate results expectation
    std::unordered_map<int, unsigned int> bin_results;
    std::unordered_map<int, unsigned int> bin_results_optional;
    // Iterate host bin
    for (unsigned int x1 = 0; x1 < 11; x1++) {
        for (unsigned int y1 = 0; y1 < 11; y1++) {
            // Solve the bin index
            const unsigned int bin_pos1[3] = {
                x1,
                y1
            };
            const unsigned int bin_index1 =
                bin_pos1[1] * 11 +
                bin_pos1[0];
            // Count our neighbours
            unsigned int count_sum = 0;
            unsigned int count_sum_optional = 0;  // NEW!
            for (int x2 = -1; x2 <= 1; x2++) {
                int bin_pos2[2] = {
                    static_cast<int>(bin_pos1[0]) + x2,
                    0
                };
                for (int y2 = -1; y2 <= 1; y2++) {
                    bin_pos2[1] = static_cast<int>(bin_pos1[1]) + y2;
                    // Ensure bin is in bounds
                    if (
                        bin_pos2[0] >= 0 &&
                        bin_pos2[1] >= 0 &&
                        bin_pos2[0] < 11 &&
                        bin_pos2[1] < 11
                        ) {
                        const unsigned int bin_index2 =
                            bin_pos2[1] * 11 +
                            bin_pos2[0];
                        count_sum += bin_counts[bin_index2];
                        count_sum_optional += bin_counts_optional[bin_index2];  // NEW!
                    }
                }
            }
            bin_results.emplace(bin_index1, count_sum);
            bin_results_optional.emplace(bin_index1, count_sum_optional);  // NEW!
        }
    }

    // Execute a single step of the model
    cuda_model.step();

    // Recover the results and check they match what was expected

    cuda_model.getPopulationData(population);
    // Validate each agent has same result
    unsigned int badCountWrong = 0;
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentInstance ai = population.getInstanceAt(i);
        unsigned int myBin = ai.getVariable<unsigned int>("myBin");
        unsigned int myResult = ai.getVariable<unsigned int>("count");
        if (ai.getVariable<unsigned int>("badCount"))
            badCountWrong++;
        EXPECT_EQ(myResult, bin_results_optional.at(myBin));  // NEW!
    }
    EXPECT_EQ(badCountWrong, 0u);
}
